#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "dvrp_algorithm.h"
#include "vrp_parse.h"

}

static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
//**********************************************************

//***********************************************************

__device__ static void d_swap(unsigned int *ar, unsigned int first,
		unsigned int second) {
	unsigned int temp = ar[first];
	ar[first] = ar[second];
	ar[second] = temp;
}

__device__ static void d_reverse(unsigned int *ar, size_t len) {
	unsigned int i, j;

	for (i = 0, j = len - 1; i < j; i++, j--) {
		d_swap(ar, i, j);
	}
}

__device__ float d_euclid_distance(int x1, int y1, int x2, int y2) {
	return sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

__device__ unsigned int d_next_permutation(unsigned int *ar, size_t len) {
	unsigned int i1, i2;
	unsigned int result = 0;

	/* Find the rightmost element that is the first in a pair in ascending order */
	for (i1 = len - 2, i2 = len - 1; ar[i2] <= ar[i1] && i1 != 0; i1--, i2--)
		;
	if (ar[i2] <= ar[i1]) {
		/* If not found, array is highest permutation */
		d_reverse(ar, len);
	} else {
		/* Find the rightmost element to the right of i1 that is greater than ar[i1] */
		for (i2 = len - 1; i2 > i1 && ar[i2] <= ar[i1]; i2--)
			;
		/* Swap it with the first one */
		d_swap(ar, i1, i2);
		/* Reverse the remainder */
		d_reverse(ar + i1 + 1, len - i1 - 1);
		result = 1;
	}
	return result;
}

__device__ float d_route_distance(unsigned int* route, size_t route_size,
		location* locations, size_t location_size, unsigned short capacity) {

	float distance = 0;
	size_t i;
	short current_load = capacity;
	location* current, *prev = NULL;

	prev = &locations[route[0]];
	current_load -= locations[route[0]].demand;
	distance += d_euclid_distance(0, 0, prev->x, prev->y);

	for (i = 1; i < route_size; i++) {
		current = &locations[route[i]];
		current_load -= locations[route[i]].demand;

		if (current_load < 0) {
			distance += d_euclid_distance(prev->x, prev->y, 0, 0); /* hardcoded awesomeness 0.0*/
			distance += d_euclid_distance(0, 0, current->x, current->y); /* more awesomeness*/
			current_load = capacity - current->demand;
		} else
			distance += d_euclid_distance(prev->x, prev->y, current->x,
					current->y);

		prev = current;
	}

	distance += d_euclid_distance(prev->x, prev->y, 0, 0);

	return distance;
}

__device__ float d_dvrp(unsigned int* permutation, vrp_info *info,
		location* locations, unsigned short blockedc) {

	//int i;
	float min_distance, distance;
	//unsigned int* min_distance_permutation;
	//hipMalloc((unsigned int*)min_distance_permutation,info.num_visits * sizeof(int));

	//hipMalloc((void** ) &kernelArray,
	//sizeof(int) * info->num_visits * WORK_SIZE));

	min_distance = d_route_distance(permutation, info->num_visits, locations,
			info->num_locations, info->capacities);
	//memcpy(min_distance_permutation, permutation,
	//sizeof(int) * info.num_visits);

	while (d_next_permutation(permutation + blockedc,
			info->num_visits - blockedc)) {
		distance = d_route_distance(permutation, info->num_visits, locations,
				info->num_locations, info->capacities);
		if (distance < min_distance) {
			min_distance = distance;
			//memcpy(min_distance_permutation, permutation,
			//sizeof(int) * info.num_visits);
			//printf("Distance: %f\n", min_distance);
			//for (i = 0; i < info.num_visits; i++)
			//	printf("%d ", min_distance_permutation[i]);
			//printf("\n");
		}
	}
	//printf("%f\n", min_distance);
	//for (i = 0; i < info.num_visits; i++)
	//printf("%d ", min_distance_permutation[i]);
	//printf("\n");
	return min_distance;

}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void kernel_dvrp(float* dst, unsigned int* permutation,
		vrp_info* info, location* locations, unsigned short blockedc,
		int problemsc) {

	float dist = 0;
	int id = blockIdx.x * WORK_SIZE + threadIdx.x;
	extern __shared__ unsigned int sh_permutation[];

	for (int i = 0; i < info->num_visits; i++) {
		sh_permutation[threadIdx.x * info->num_visits + i] =
				permutation[id + i];
	}
	//sh_permutation=(unsigned int*) malloc(sizeof(unsigned int)*info->num_visits*WORK_SIZE);
	//memcpy(sh_permutation,permutation + blockIdx.x * info->num_visits*WORK_SIZE,info->num_visits*WORK_SIZE);

	//printf("threadidx.x  = %d\n", threadIdx.x);
	/*
	 if (threadIdx.x == 1) {
	 printf("threadidx.x  = %d\n", threadIdx.x);
	 printf("info num_visits = %d\n", info->num_visits);
	 for (int i = 0; i < 256; i++) {
	 for (int j = 0; j < 12; j++)
	 printf("%d ", *(permutation + i * info->num_visits + j));
	 printf("\n");
	 }
	 }
	 */
	//unsigned int *idata = ((unsigned int*) permutation
	//+ threadIdx.x * info.num_visits);
	//printf("threadidx.x  = %d\n",threadIdx.x);
	if (id < problemsc) {
		dist = d_dvrp(sh_permutation + threadIdx.x * info->num_visits, info,
				locations, blockedc);

		dst[id] = dist;
		//dst[id] ma poprawne wartosci
		//printf("problem = %d\t distance = %f \t dst[id] =  %f\n", id, dist,dst[id]);
	}
	/*
	 if (id == 0) {
	 for (int i = 0; i < problemsc; i++) {
	 printf("problem = %d\t distance = %f \n", i, dst[i]);
	 }

	 }
	 __syncthreads();
	 */
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	unsigned int* kernelArray = NULL;
	unsigned int i = 1, k, n;

	unsigned int* permutation;
	unsigned int* problems;
	float* d_odata;
	float* h_odata;
	unsigned int problemsc = 1;

	vrp_info* info, *d_info;
	location* locations, *d_locations;

	//divide

	n = WORK_SIZE;
	parse_vrp("", &locations, &info);
	k = info->num_visits;
	while (k < n)
		k *= k - i++;
	k = i;
	permutation = (unsigned int*) malloc(sizeof(int) * info->num_visits);
	for (i = 0; i < info->num_visits; i++)
		permutation[i] = i + 1;

	for (i = 0; i < k; i++)
		problemsc *= info->num_visits - i;

	printf("problemsc = %d\n", problemsc);

	problems = (unsigned int*) malloc(sizeof(int*) * problemsc);

	problems = divide(permutation, info->num_visits, k);

	printf("k = %d \t n = %d \n", k, n);

	for (int i = 0; i < problemsc; i++) {
		printf("problems[%d] = ", i);
		for (int j = 0; j < info->num_visits; j++) {
			printf("%d ", *(problems + i * info->num_visits + j));
		}
		printf("\n");
	}

	//solve

	printf("kernelArray hipMalloc\n");
	//for (int i = 0; i < problemsc / WORK_SIZE; i++) {
	CUDA_CHECK_RETURN(
			hipMalloc((unsigned int** ) &kernelArray,
					sizeof(int) * info->num_visits * WORK_SIZE));
	printf("d_info hipMalloc\n");
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_info, sizeof(vrp_info)));
	printf("d_locations hipMalloc\n");
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &d_locations,
					sizeof(location) * info->num_locations));

	printf("d_odata hipMalloc\n");
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_odata, sizeof(float) * problemsc));

	printf("kernelArray from problems hipMemcpy\n");
	CUDA_CHECK_RETURN(
			hipMemcpy(kernelArray, problems,
					sizeof(unsigned int) * info->num_visits * WORK_SIZE,
					hipMemcpyHostToDevice));

	printf("d_info from info hipMemcpy\n");
	CUDA_CHECK_RETURN(
			hipMemcpy(d_info, info, sizeof(vrp_info), hipMemcpyHostToDevice));

	printf("d_locations from locations hipMemcpy\n");
	CUDA_CHECK_RETURN(
			hipMemcpy(d_locations, locations,
					sizeof(location) * info->num_locations,
					hipMemcpyHostToDevice));

	printf("sharedMemory size\n");
	unsigned int sharedMemory = WORK_SIZE * info->num_visits
			* sizeof(unsigned int) + sizeof(vrp_info)
			+ sizeof(location) * info->num_locations;

	printf("kernel call\n");
	kernel_dvrp<<<6, WORK_SIZE, sharedMemory>>>(d_odata, kernelArray, d_info,
			d_locations, k, problemsc);

	//printf("out of kernel10\n");
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	//CUDA_CHECK_RETURN(hipDeviceSynchronize());
	printf("get cudaerrors\n");
	CUDA_CHECK_RETURN(hipGetLastError());

	printf("h_odata malloc\n");
	h_odata = (float*) malloc(sizeof(float) * problemsc);
	printf("h_odata from d_odata hipMemcpy\n");
	CUDA_CHECK_RETURN(
			hipMemcpy(h_odata, d_odata, sizeof(float) * problemsc,
					hipMemcpyDeviceToHost));

	hipFree(d_odata);

	printf("print h_odata : \n");
	for (int j = 0; j < problemsc; j++) {
		printf("h_odata[%d] =  %f\n", j, h_odata[j]);
	}

	printf("end of program\n");
	return EXIT_SUCCESS;
}
